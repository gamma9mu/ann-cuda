#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "backprop.h"
}

__device__ inline float
sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

/* Performs backpropagation learning on a feed forward neural netowrk.
 * data         column-major, 2-d array of inputs (each of INPUT_SIZE length)
 * count        the total number of items in data
 * expected     column-major, 2-d array of expected output values (each of
 *              OUTPUT_SIZE length)
 * w_ih         column-major weight matrix for the hidden layer
 * theta_h      activation weights of the hidden layer
 * w_ho         column-major weight matrix for the output layer
 * theta_o      activation weights of the output layer
 * rate         learning rate of the ANN (a float between 0 and 1)
 *
 * The updated weight matrices will be copied over their previous values in
 * device global memory after each of the items in data has been processed.
 *
 * This kernel should be called with 1 block, having a single dimension of
 * threads, where the number of threads is the maximum of the number of hidden
 * layer neurons and the number of output layer neurons.
 */
__global__ void
backprop(float *data, int count, float *expected,
        float *w_ih, float *theta_h, float *w_ho, float *theta_o,
        float rate)
{
    /* Hidden layer weights */
    __shared__ float w_hid[HIDDEN_SIZE][INPUT_SIZE];
    __shared__ float th_h[HIDDEN_SIZE];

    /* Output layer weights */
    __shared__ float w_out[OUTPUT_SIZE][HIDDEN_SIZE];
    __shared__ float th_o[HIDDEN_SIZE];

    /* Layer output values */
    __shared__ float hid[HIDDEN_SIZE];
    __shared__ float out[OUTPUT_SIZE];
    __shared__ float err[OUTPUT_SIZE];
    __shared__ float delta[OUTPUT_SIZE];

    /* Input data */
    __shared__ float input[INPUT_SIZE];

    int tx = threadIdx.x;

    /* Load the hidden layer's theta values and weight matrix. */
    if (tx < HIDDEN_SIZE) {
        th_h[tx] = theta_h[tx];
        for (int i = 0; i < INPUT_SIZE; ++i)
            w_hid[tx][i] = w_ih[tx + (INPUT_SIZE * i)];
    }

    /* Load the output layer's theta values and weight matrix. */
    if (tx < OUTPUT_SIZE) {
        th_o[tx] = theta_o[tx];
        for (int i = 0; i < HIDDEN_SIZE; ++i)
            w_out[tx][i] = w_ho[tx + (OUTPUT_SIZE * i)];
    }

    /* Process each piece of input data. */
    for (int i = 0; i < count; ++i) {
        /* Load the data item. */
        if (tx < INPUT_SIZE)
            input[tx] = data[tx + (i * INPUT_SIZE)];

        __syncthreads();
        
        /* Propagate through the hidden layer. */
        if (tx < HIDDEN_SIZE) {
            hid[tx] = 0.0f;
            for (int j = 0; j < INPUT_SIZE; ++j)
                hid[tx] += input[j] * w_hid[tx][j];
            hid[tx] -= th_h[tx];
            hid[tx] = sigmoid(hid[tx]);
        }
        
        __syncthreads();

        /* Propagate through the output layer. */
        if (tx < OUTPUT_SIZE) {
            out[tx] = 0.0f;
            for (int j = 0; j < HIDDEN_SIZE; ++j)
                out[tx] += hid[j] * w_out[tx][j];
            out[tx] -= th_o[tx];
            out[tx] = sigmoid(out[tx]);
        }

        __syncthreads();

        /* Backpropagation starts here. */

        /* Calculate the error deltas for the output layer. */
        if (tx < OUTPUT_SIZE) {
            err[tx] = expected[tx + (OUTPUT_SIZE * i)] - out[tx];
            delta[tx] = out[tx] * (1 - out[tx]) * err[tx];
        }

        __syncthreads();

        /* Calculate the error deltas for the hidden layer and update the
         * weights. */
        if (tx < HIDDEN_SIZE) {
            float hdelta = 0.0f;
            for (int j = 0; j < OUTPUT_SIZE; ++j)
                hdelta += delta[j] * hid[tx];
            hdelta *= rate;
            for (int j = 0; j < INPUT_SIZE; ++j)
                w_hid[tx][j] += hdelta * input[j];
        }

        __syncthreads();

        /* Update the input weights of the output layer.  This section cannot be
         * combined above.  The changes would affect the hidden layer updates.
         */
        if (tx < OUTPUT_SIZE)
            for (int j = 0; j < HIDDEN_SIZE; ++j)
                w_out[tx][j] += rate * hid[j] * delta[tx];

        __syncthreads();
    }

    /* Copy the hidden layer's weight matrix out to global memory. */
    if (tx < HIDDEN_SIZE) {
        for (int i = 0; i < INPUT_SIZE; ++i)
             w_ih[tx + (INPUT_SIZE * i)] = w_hid[tx][i];
    }

    /* Copy the output layer's weight matrix out to global memory. */
    if (tx < OUTPUT_SIZE) {
        for (int i = 0; i < HIDDEN_SIZE; ++i)
             w_ho[tx + (OUTPUT_SIZE * i)] = w_out[tx][i];
    }
}

/* Copies data to device global memory from host memory
 * data         column-major, 2-d array of inputs(each of INPUT_SIZE length
 * count        the total number of items in data
 * expected     column-major, 2-d array of expected output values (each of
 *              OUTPUT_SIZE length)
 * w_ih         column-major weight matrix for the hidden layer
 * theta_h      activation weights of the hidden layer
 * w_ho         column-major weight matrix for the output layer
 * theta_o      activation weights of the output layer
 * rate         learning rate of the ANN (a float between 0 and 1)
 *
 * This wrapper calls the backprop kernel with 1 block
 */

void backprop_wrapper(float *data, int count, float *expected, 
        float *w_ih, float *theta_h, float *w_ho, float *theta_o,
        float rate){
    size_t input_size = (count * INPUT_SIZE) * sizeof(float);
    //size_t output_data_size = (count * OUTPUT_SIZE) * sizeof(float);
    size_t hidden_size = (count * HIDDEN_SIZE) * sizeof(float);
    size_t output_size = (count * OUTPUT_SIZE) * sizeof(float);
    
    float* d_data;
    hipMalloc(&d_data, input_size);

    float* d_expected;
    hipMalloc(&d_expected, output_size);

    float* d_w_ih;
    hipMalloc(&d_w_ih, hidden_size);

    float* d_theta_h;
    hipMalloc(&d_theta_h, hidden_size);

    float* d_w_ho;
    hipMalloc(&d_w_ho, output_size);

    float* d_theta_o;
    hipMalloc(&d_theta_o, output_size);

    hipMemcpy(d_data, data, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_expected, expected, output_size, hipMemcpyHostToDevice);
    hipMemcpy(d_w_ih, w_ih, hidden_size, hipMemcpyHostToDevice);
    hipMemcpy(d_theta_h, theta_h, hidden_size, hipMemcpyHostToDevice);
    hipMemcpy(d_w_ho, w_ho, output_size, hipMemcpyHostToDevice);
    hipMemcpy(d_theta_o, theta_o, output_size, hipMemcpyHostToDevice);

    int ThreadsPerBlock;
    if(OUTPUT_SIZE > HIDDEN_SIZE)
        ThreadsPerBlock = OUTPUT_SIZE;
    else
        ThreadsPerBlock = HIDDEN_SIZE;

    backprop<<<1, ThreadsPerBlock>>>(d_data, count, d_expected, d_w_ih,
            d_theta_h, d_w_ho, d_theta_o, rate);

    hipMemcpy(w_ho, d_w_ho, output_size, hipMemcpyDeviceToHost);
    hipMemcpy(w_ih, d_w_ih, hidden_size, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_expected);
    hipFree(d_w_ih);
    hipFree(d_theta_h);
    hipFree(d_w_ho);
    hipFree(d_theta_o);
}

/* Evaluates an ANN's sum of squared errors.
 * data         column-major, 2-d array of inputs (each of INPUT_SIZE length)
 * count        the total number of items in data
 * expected     column-major, 2-d array of expected output values (each of
 *              OUTPUT_SIZE length)
 * w_ih         column-major weight matrix for the hidden layer
 * theta_h      activation weights of the hidden layer
 * w_ho         column-major weight matrix for the output layer
 * theta_o      activation weights of the output layer
 * sse          address in global memory to store the SSE into
 *
 * This kernel should be called with 1 block, having a single dimension of
 * threads, where the number of threads is the maximum of the number of hidden
 * layer neurons and the number of output layer neurons.
 */
__global__ void
evaluate(float *data, int count, float *expected,
        float *w_ih, float *theta_h, float *w_ho, float *theta_o,
        float *sse)
{
    /* Hidden layer weights */
    __shared__ float w_hid[HIDDEN_SIZE][INPUT_SIZE];
    __shared__ float th_h[HIDDEN_SIZE];

    /* Output layer weights */
    __shared__ float w_out[OUTPUT_SIZE][HIDDEN_SIZE];
    __shared__ float th_o[HIDDEN_SIZE];

    /* Layer output values */
    __shared__ float hid[HIDDEN_SIZE];
    __shared__ float out[OUTPUT_SIZE];
    __shared__ float errors[OUTPUT_SIZE];

    /* Input data */
    __shared__ float input[INPUT_SIZE];

    int tx = threadIdx.x;

    /* Load the hidden layer's theta values and weight matrix. */
    if (tx < HIDDEN_SIZE) {
        th_h[tx] = theta_h[tx];
        for (int i = 0; i < INPUT_SIZE; ++i)
            w_hid[tx][i] = w_ih[tx + (INPUT_SIZE * i)];
    }

    /* Load the output layer's theta values and weight matrix. */
    if (tx < OUTPUT_SIZE) {
        th_o[tx] = theta_o[tx];
        for (int i = 0; i < HIDDEN_SIZE; ++i)
            w_out[tx][i] = w_ho[tx + (OUTPUT_SIZE * i)];
        errors[tx] = 0.0f;
    }

    /* Process each piece of input data. */
    for (int i = 0; i < count; ++i) {
        /* Load the data item. */
        if (tx < INPUT_SIZE)
            input[tx] = data[tx + (i * INPUT_SIZE)];

        __syncthreads();
        
        /* Propagate through the hidden layer. */
        if (tx < HIDDEN_SIZE) {
            hid[tx] = 0.0f;
            for (int j = 0; j < INPUT_SIZE; ++j)
                hid[tx] += input[j] * w_hid[tx][j];
            hid[tx] -= th_h[tx];
            hid[tx] = sigmoid(hid[tx]);
        }
        
        __syncthreads();

        /* Propagate through the output layer. */
        if (tx < OUTPUT_SIZE) {
            out[tx] = 0.0f;
            for (int j = 0; j < HIDDEN_SIZE; ++j)
                out[tx] += hid[j] * w_out[tx][j];
            out[tx] -= th_o[tx];
            out[tx] = sigmoid(out[tx]);
        }

        __syncthreads();

        /* Track each output neuron's squared errors. */
        if (tx < OUTPUT_SIZE) {
            float error = expected[tx + (OUTPUT_SIZE * i)] - out[tx];
            errors[tx] += error * error;
        }

        __syncthreads();
    }

    /* Sum individual output neuron's SSE and write to global memory. */
    if (tx == 0) {
        float errsum = 0.0f;
        for (int i = 0; i < OUTPUT_SIZE; ++i)
            errsum += errors[i];
        *sse = errsum;
    }
}

/* Copies over data to device global memory for evaluate
 * data         column-major, 2-d array of inputs (each of INPUT_SIZE length)
 * count        the total number of items in data
 * expected     column-major, 2-d array of expected output values (each of
 *              OUTPUT_SIZE length)
 * w_ih         column-major weight matrix for the hidden layer
 * theta_h      activation weights of the hidden layer
 * w_ho         column-major weight matrix for the output layer
 * theta_o      activation weights of the output layer
 * sse          address in global memory to store the SSE into
 *
 * Calls the evaluate kernel with one block, having a single dimension of
 * threads, where the number of threads is the maximum of the number of hidden
 * layer neurons and the number of output layer neurons.
 */

/* Runs an ANN on a series of inputs.
 * data         column-major, 2-d array of inputs (each of INPUT_SIZE length)
 * count        the total number of items in data
 * w_ih         column-major weight matrix for the hidden layer
 * theta_h      activation weights of the hidden layer
 * w_ho         column-major weight matrix for the output layer
 * theta_o      activation weights of the output layer
 * output       column-major, 2-d array to hold the output values (each of
 *              OUTPUT_SIZE length)
 *
 * This kernel should be called with 1 block, having a single dimension of
 * threads, where the number of threads is the maximum of the number of hidden
 * layer neurons and the number of output layer neurons.
 */
__global__ void
run(float *data, int count, float *expected,
        float *w_ih, float *theta_h, float *w_ho, float *theta_o,
        float *output)
{
    /* Hidden layer weights */
    __shared__ float w_hid[HIDDEN_SIZE][INPUT_SIZE];
    __shared__ float th_h[HIDDEN_SIZE];

    /* Output layer weights */
    __shared__ float w_out[OUTPUT_SIZE][HIDDEN_SIZE];
    __shared__ float th_o[HIDDEN_SIZE];

    /* Layer output values */
    __shared__ float hid[HIDDEN_SIZE];
    __shared__ float out[OUTPUT_SIZE];

    /* Input data */
    __shared__ float input[INPUT_SIZE];

    int tx = threadIdx.x;

    /* Load the hidden layer's theta values and weight matrix. */
    if (tx < HIDDEN_SIZE) {
        th_h[tx] = theta_h[tx];
        for (int i = 0; i < INPUT_SIZE; ++i)
            w_hid[tx][i] = w_ih[tx + (INPUT_SIZE * i)];
    }

    /* Load the output layer's theta values and weight matrix. */
    if (tx < OUTPUT_SIZE) {
        th_o[tx] = theta_o[tx];
        for (int i = 0; i < HIDDEN_SIZE; ++i)
            w_out[tx][i] = w_ho[tx + (OUTPUT_SIZE * i)];
    }

    /* Process each piece of input data. */
    for (int i = 0; i < count; ++i) {
        /* Load the data item. */
        if (tx < INPUT_SIZE)
            input[tx] = data[tx + (i * INPUT_SIZE)];

        __syncthreads();
        
        /* Propagate through the hidden layer. */
        if (tx < HIDDEN_SIZE) {
            hid[tx] = 0.0f;
            for (int j = 0; j < INPUT_SIZE; ++j)
                hid[tx] += input[j] * w_hid[tx][j];
            hid[tx] -= th_h[tx];
            hid[tx] = sigmoid(hid[tx]);
        }
        
        __syncthreads();

        /* Propagate through the output layer and write the results to device
         * global memory. */
        if (tx < OUTPUT_SIZE) {
            out[tx] = 0.0f;
            for (int j = 0; j < HIDDEN_SIZE; ++j)
                out[tx] += hid[j] * w_out[tx][j];
            out[tx] -= th_o[tx];
            output[tx + (OUTPUT_SIZE * i)] = sigmoid(out[tx]);
        }

        __syncthreads();
    }
}
